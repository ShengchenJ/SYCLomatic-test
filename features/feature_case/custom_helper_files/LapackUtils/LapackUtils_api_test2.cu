// ===------ LapackUtils_api_test2.cu --------------------- *- CUDA -* ---=== //
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
// ===--------------------------------------------------------------------=== //

// TEST_FEATURE: LapackUtils_hegvd


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  float2 *a_s, *b_s, *work_s;
  float *w_s;
  int lwork_s;
  int *devInfo;
  hipsolverDnChegvd(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_s, 3, b_s, 3, w_s, work_s, lwork_s, devInfo);
  return 0;
}
