#include <stdio.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

// CUDA kernel to square a half-precision floating-point number
__global__ void squareKernel(__half *d_in, __half *d_out) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    d_out[idx] = __hmul(d_in[idx], d_in[idx]);
}

int main() {
    // Initialize half-precision floating-point value
    __half h_in = __float2half(3.0f);
    __half h_out;

    // Allocate memory on the device
    __half *d_in, *d_out;
    hipMalloc((void **)&d_in, sizeof(__half));
    hipMalloc((void **)&d_out, sizeof(__half));

    // Copy the value from host to device
    hipMemcpy(d_in, &h_in, sizeof(__half), hipMemcpyHostToDevice);

    // Launch the kernel to square the value
    squareKernel<<<1, 1>>>(d_in, d_out);

    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "squareKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching squareKernel!\n", cudaStatus);
    }

    // Copy the result back to the host
    hipMemcpy(&h_out, d_out, sizeof(__half), hipMemcpyDeviceToHost);

    // Check the result
    float result = __half2float(h_out);
    printf("The square of 3.0 is: %f\n", result);

    // Free device memory
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}

