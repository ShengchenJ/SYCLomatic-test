#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel to add a float value with itself
__global__ void addKernel(float *d_in, float *d_out) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    d_out[idx] = d_in[idx];
}

int main() {
    // Initialize float value
    float a = 3.0;
    float f_in = a;
    float f_out;

    // Allocate memory on the device
    float *d_in, *d_out;
    hipMalloc((void **)&d_in, sizeof(float));
    hipMalloc((void **)&d_out, sizeof(float));

    // Copy the value from host to device
    hipMemcpy(d_in, &f_in, sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel to add the value with itself
    addKernel<<<1, 1>>>(d_in, d_out);

    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    // Copy the result back to the host
    hipMemcpy(&f_out, d_out, sizeof(float), hipMemcpyDeviceToHost);

    // Check the result
    printf("The sum of 3.0 and 3.0 is: %f\n", f_out);

    // Free device memory
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}

