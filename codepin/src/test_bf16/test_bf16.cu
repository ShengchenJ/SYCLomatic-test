#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>

// CUDA kernel to add a bfloat16 value with itself
__global__ void addKernel(hip_bfloat16 *d_in, hip_bfloat16 *d_out) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    d_out[idx] = d_in[idx];
}

int main() {
    // Initialize bfloat16 value
        float a= 3.0;
    hip_bfloat16 h_in = __float2bfloat16(a);
    hip_bfloat16 h_out;

    // Allocate memory on the device
    hip_bfloat16 *d_in, *d_out;
    hipMalloc((void **)&d_in, sizeof(hip_bfloat16));
    hipMalloc((void **)&d_out, sizeof(hip_bfloat16));

    // Copy the value from host to device
    hipMemcpy(d_in, &h_in, sizeof(hip_bfloat16), hipMemcpyHostToDevice);

    // Launch the kernel to add the value with itself
    addKernel<<<1, 1>>>(d_in, d_out);

    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    // Copy the result back to the host
    hipMemcpy(&h_out, d_out, sizeof(hip_bfloat16), hipMemcpyDeviceToHost);

    // Check the result
    float result = __bfloat162float(h_out);
    printf("The sum of 3.0 and 3.0 is: %f\n", result);

    // Free device memory
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}

